#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <chrono>
#include <random>

#include <SFML/Graphics.hpp>

#include "obj_read.cu"


const int SCENE_NUM = 0;

const Vec3 SKY_COLOUR(0.8, 1, 1);

const std::string CAPTION = "ray tracer";


int get_time() {
    //get ms since epoch
    auto clock = std::chrono::system_clock::now();
    auto duration = clock.time_since_epoch();
    int time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

    return time;
}


float host_rng(float min, float max) {
    std::random_device rand_device_seed;
    std::mt19937 gen(rand_device_seed());
    
    std::uniform_real_distribution<> dis(0.0, 1.0);

    float normalised = dis(gen);

    return min + normalised * (max - min);
}


class ImageTexture {
    public:
        std::string PARSED_TEXTURE_FILENAME = "textures/parsed_textures.txt";

        ImageTexture(std::string filename) {
            parse_file(filename);
        }

        Texture get_device_texture() {
            return Texture::create_image(width, height, rgb_values);
        }

    private:
        int width;
        int height;

        std::vector<Vec3> rgb_values;

        void parse_file(std::string filename) {
            std::vector<std::string> lines = read_file(PARSED_TEXTURE_FILENAME);

            for (int i = 0; i < lines.size(); i++) {
                if (lines[i] == filename) {
                    width = std::stoi(lines[i + 1]);
                    height = std::stoi(lines[i + 2]);
                    rgb_values = parse_rgb_values(lines[i + 3]);

                    return;
                }
            }

            //if we have not yet returned from the function, the file was not found
            throw std::runtime_error("Image file not found.\n");
        }

        std::vector<Vec3> parse_rgb_values(std::string rgb_string) {
            std::vector<std::string> splitted = split_string(rgb_string, ' ');

            std::vector<Vec3> parsed_rgb;

            //NOTE: the last character will just be "", so loop to the last but one character
            for (int i = 0; i < splitted.size() - 1; i += 3) {
                float r = std::stof(splitted[i]);
                float g = std::stof(splitted[i + 1]);
                float b = std::stof(splitted[i + 2]);

                parsed_rgb.push_back(Vec3(r, g, b));
            }

            return parsed_rgb;
        }
};


class SceneObjects {
    public:
        AllObjects gpu_struct;

        bool use_sky = true;

        SceneObjects(int test_scene) {
            switch (test_scene) {
                case 0:
                    monkey_test_scene();
                    break;
                case 1:
                    reflection_test_scene();
                    break;
                case 2:
                    texture_test_scene();
                    break;
                case 3:
                    refract_test_scene();
                    break;
                case 4:
                    rand_sphere_test_scene();
                    break;
                default:
                    throw std::domain_error("Test scene must be number between 0 and 3 (inclusive).\n");
            }

            gpu_struct = create_gpu_struct();
        }

    private:
        std::vector<Object> objects;

        Object create_mesh(ObjFileMesh obj, Material mat) {
            //parse the object file mesh faces into triangles and add create a mesh instance that can be used on the gpu
            std::vector<Triangle> triangles;

            for (std::vector<float3> face : obj.faces) {
                if (face.size() == 3) {
                    Triangle tri(Vec3(face[0]), Vec3(face[1]), Vec3(face[2]), mat);
                    triangles.push_back(tri);
                } else if (face.size() == 4) {
                    Quad quad(Vec3(face[0]), Vec3(face[1]), Vec3(face[2]), Vec3(face[3]), mat);
                    
                    triangles.push_back(quad.t1);
                    triangles.push_back(quad.t2);
                } else {
                    throw std::logic_error("Only triangle or quad meshes are supported.\n");
                }
            }

            ReadOnlyDeviceArray<Triangle> device_array(triangles);

            return Object::create_mesh(triangles, device_array.device_pointer, mat);
        }

        void monkey_test_scene() {
            //setup simple test scene with a cornell box, suzanne mesh and sphere
            /*
            Texture monkey_tex = Texture::create_const_colour(Vec3(1, 1, 1));
            Material monkey_mat = Material::create_standard(monkey_tex, 0);

            ObjFileMesh m("models/low_poly_monkey.obj");
            m.enlarge(0.3);
            m.rotate(0, 2.3, 0);
            m.translate(0.1, -0.1, 1.6);

            Object monkey_mesh = create_mesh(m, monkey_mat);
            objects.push_back(monkey_mesh);

            return;
            */


            create_cornell_box(Vec3(-0.5, 0.5, 1.2), 1, 1, 1, 0.5);

            Texture monkey_tex = Texture::create_const_colour(Vec3(1, 1, 1));
            Material monkey_mat = Material::create_standard(monkey_tex, 0);

            ObjFileMesh m("models/low_poly_monkey.obj");
            m.enlarge(0.3);
            m.rotate(0, 2.3, 0);
            m.translate(0.1, -0.1, 1.6);

            Object monkey_mesh = create_mesh(m, monkey_mat);
            objects.push_back(monkey_mesh);

            Texture sphere_tex = Texture::create_const_colour(Vec3(0.8, 0.8, 0.8));
            Material sphere_mat = Material::create_standard(sphere_tex, 1);
            Object sphere = Object::create_sphere(Vec3(-0.25, -0.25, 1.95), 0.25, sphere_mat);

            objects.push_back(sphere);
        }

        void reflection_test_scene() {
            //simple test scene with spheres of different smoothness values
            create_cornell_box(Vec3(-0.5, 0.5, 1.2), 1, 1, 1, 0.5);

            Texture sphere_tex = Texture::create_const_colour(Vec3(1, 1, 1));

            Material a = Material::create_standard(sphere_tex, 0);
            Material b = Material::create_standard(sphere_tex, 0.33);
            Material c = Material::create_standard(sphere_tex, 0.66);
            Material d = Material::create_standard(sphere_tex, 1);

            objects.push_back(Object::create_sphere(Vec3(-0.2, 0.2, 1.7), 0.15, a));
            objects.push_back(Object::create_sphere(Vec3(0.2, 0.2, 1.7), 0.15, b));
            objects.push_back(Object::create_sphere(Vec3(-0.2, -0.2, 1.7), 0.15, c));
            objects.push_back(Object::create_sphere(Vec3(0.2, -0.2, 1.7), 0.15, d));
        }

        void texture_test_scene() {
            //test scene with spheres with tetxtures
            create_cornell_box(Vec3(-0.5, 0.5, 1.2), 1, 1, 1, 0.5);

            ImageTexture earth("earth.png");
            Material earth_mat = Material::create_standard(earth.get_device_texture(), 0);

            objects.push_back(Object::create_sphere(Vec3(0, 0, 1.7), 0.25, earth_mat));

            Texture tri_tex = Texture::create_checkerboard(Vec3(1, 1, 1), Vec3(0, 0, 0), 4);
            Material tri_mat = Material::create_standard(tri_tex, 0);

            Object t1 = Object::create_triangle(Vertex{Vec3(0.1, 0, 1.7), Vec2(0, 0)}, Vertex{Vec3(0.6, 0.5, 1.9), Vec2(0, 1)}, Vertex{Vec3(0.8, 0.4, 2), Vec2(1, 1)}, tri_mat);

            objects.push_back(t1);
        }

        void refract_test_scene() {
            create_cornell_box(Vec3(-0.5, 0.5, 1.2), 1, 1, 1, 0.5);

            Texture refract_tex = Texture::create_const_colour(Vec3(1, 1, 1));
            Material refract_mat = Material::create_refractive(refract_tex, 1.5);

            objects.push_back(Object::create_sphere(Vec3(0, -0.1, 1.7), 0.3, refract_mat));
        }

        void rand_sphere_test_scene() {
            //final scene from https://raytracing.github.io/books/RayTracingInOneWeekend.html#wherenext?/afinalrender
            int num_spheres = 100;

            float floor_y = -1;
            float floor_width = 10;
            float floor_depth = 10;

            for (int i = 0; i < num_spheres; i++) {
                Material material;
                Texture tex = Texture::create_const_colour(Vec3(host_rng(0, 1), host_rng(0, 1), host_rng(0, 1)));

                float mat_num = host_rng(0, 1);

                if (mat_num < 0.3) {
                    float smoothness = host_rng(0, 1);

                    material = Material::create_standard(tex, smoothness); 
                } else if (mat_num < 0.6) {
                    float refract_inx = host_rng(0.5, 2);

                    material = Material::create_refractive(tex, refract_inx);
                }

                float radius = host_rng(0.1, 0.5);

                Vec3 center(host_rng(-floor_width / 2, floor_width / 2), floor_y + radius, host_rng(0, floor_depth));

                objects.push_back(Object::create_sphere(center, radius, material));
            }

            Texture floor_tex = Texture::create_checkerboard(Vec3(0.7, 0.7, 0.7), Vec3(0.4, 0.4, 0.4), 10);
            Material floor_mat = Material::create_standard(floor_tex, 0);

            objects.push_back(Object::create_quad(Vec3(-floor_width / 2, floor_y, 0), Vec3(floor_width / 2, floor_y, 0), Vec3(floor_width / 2, floor_y, floor_depth), Vec3(-floor_width / 2, floor_y, floor_depth), floor_mat));
        }

        void create_cornell_box(Vec3 tl_near_pos, float width, float height, float depth, float light_width) {
            use_sky = false;

            Texture floor_tex = Texture::create_checkerboard(Vec3(0.1, 0.8, 0.1), Vec3(0.1, 0.5, 0.1), 8);
            Texture l_wall_tex = Texture::create_const_colour(Vec3(1, 0.2, 0.2));
            Texture r_wall_tex = Texture::create_const_colour(Vec3(0.3, 0.3, 1));
            Texture back_tex = Texture::create_const_colour(Vec3(0.2, 0.2, 0.2));
            Texture roof_tex = Texture::create_const_colour(Vec3(0.9, 0.9, 0.9));
            Texture front_tex = Texture::create_const_colour(Vec3(1, 1, 1));

            Material floor = Material::create_standard(floor_tex, 0);
            Material l_wall = Material::create_standard(l_wall_tex, 0);
            Material r_wall = Material::create_standard(r_wall_tex, 0);
            Material back = Material::create_standard(back_tex, 0);
            Material roof = Material::create_standard(roof_tex, 0);
            Material front = Material::create_standard(front_tex, 0);

            //offset vectors
            Vec3 w(width, 0, 0);
            Vec3 h(0, height, 0);
            Vec3 d(0, 0, depth);

            objects.push_back(Object::create_quad(tl_near_pos - h, tl_near_pos - h + w, tl_near_pos - h + w + d, tl_near_pos - h + d, floor));
            objects.push_back(Object::create_quad(tl_near_pos, tl_near_pos - h, tl_near_pos - h + d, tl_near_pos + d, l_wall));
            objects.push_back(Object::create_quad(tl_near_pos + w, tl_near_pos + w - h, tl_near_pos + w - h + d, tl_near_pos + w + d, r_wall));
            objects.push_back(Object::create_quad(tl_near_pos + d, tl_near_pos + w + d, tl_near_pos + w - h + d, tl_near_pos - h + d, back));
            objects.push_back(Object::create_quad(tl_near_pos, tl_near_pos + d, tl_near_pos + w + d, tl_near_pos + w, roof));
            objects.push_back(Object::create_one_way_quad(tl_near_pos, tl_near_pos + w, tl_near_pos + w - h, tl_near_pos - h, false, front));  //front wall is one way so we can see through it

            //add the light
            Material light_mat = Material::create_emissive(Vec3(1, 1, 1), 6);

            Vec3 light_tl_near_pos(tl_near_pos.x + width / 2 - light_width / 2, tl_near_pos.y, tl_near_pos.z + depth / 2 - light_width / 2);  //ensure light is in center of roof
            Object light = Object::create_cuboid(light_tl_near_pos, light_width, 0.04, light_width, light_mat);

            objects.push_back(light);
        }

        AllObjects create_gpu_struct() {
            int num_meshes = objects.size();
            ReadOnlyDeviceArray<Object> array(objects);

            return AllObjects{array.device_pointer, num_meshes};
        }
};


class RenderSettings {
    public:
        RenderData gpu_struct;

        RenderSettings(bool use_sky) {
            assign_default(use_sky);

            gpu_struct = RenderData{rays_per_pixel, reflect_limit, antialias, sky_colour};
        }

    private:
        int reflect_limit;
        int rays_per_pixel;

        bool static_scene;
        bool antialias;

        Vec3 sky_colour;

        void assign_default(bool use_sky) {
            //these settings can be changed
            reflect_limit = 5;
            rays_per_pixel = 100;

            antialias = true;

            if (use_sky) {
                sky_colour = SKY_COLOUR;
            } else {
                sky_colour = Vec3(0, 0, 0);
            }
        }
};


std::vector<float> get_pixel_colours(VariableRenderData *render_data) {
    //get the pixel colours from the raytracer
    int time = get_time();
    render(render_data, time);  //will update render_data.previous_render

    return render_data->previous_render;
}


std::vector<sf::Uint8> parse_pixel_colours(std::vector<float> pixel_colours) {
    //turn array of rgb floats between 0 and 1 into something that can be drawn
    std::vector<sf::Uint8> parsed_pixel_colours(SCREEN_WIDTH * SCREEN_HEIGHT * 4);

    for (int x = 0; x < SCREEN_WIDTH; x++) {
        for (int y = 0; y < SCREEN_HEIGHT; y++) {
            int pixel_colour_inx = (y * SCREEN_WIDTH + x) * 3;
            int result_inx = (y * SCREEN_WIDTH + x) * 4;

            //add the rgb colours
            for (int i = 0; i < 3; i++) {
                int colour = pixel_colours[pixel_colour_inx + i] * 255;

                if (colour > 255) {
                    colour = 255;
                } else if (colour < 0) {
                    colour = 0;
                }

                sf::Uint8 converted_colour = colour;
                parsed_pixel_colours[result_inx + i] = converted_colour;
            }

            parsed_pixel_colours[result_inx + 3] = 255;  //add alpha
        }
    }

    return parsed_pixel_colours;
}


void draw_screen(sf::RenderWindow *window, std::vector<float> pixel_colours) {
    std::vector<sf::Uint8> rgba_colours = parse_pixel_colours(pixel_colours);

    //create a texture continaing the pixel colours
    sf::Texture texture;
    texture.create(SCREEN_WIDTH, SCREEN_HEIGHT);
    texture.update(rgba_colours.data());

    sf::Sprite sprite(texture);

    window->draw(sprite);
    window->display();
}


void init() {
    Camera camera;

    camera.assign_constant_mem();
    
    SceneObjects mesh_data(SCENE_NUM);
    RenderSettings render_data(mesh_data.use_sky);

    allocate_constant_mem(render_data.gpu_struct, mesh_data.gpu_struct);
}


int main() {
    init();

    sf::VideoMode dims(SCREEN_WIDTH, SCREEN_HEIGHT);
    sf::RenderWindow window(dims, CAPTION);

    VariableRenderData render_data{0, std::vector<float>(PIXEL_ARRAY_LEN, 0)};

    int start_time = get_time();

    while (window.isOpen()) {
        //check if the window has been closed
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
            }
        }

        std::vector<float> pixel_colours = get_pixel_colours(&render_data);
        draw_screen(&window, pixel_colours);

        int elapsed = get_time() - start_time;
        float fps = 1000 / static_cast<float>(elapsed);
        start_time = get_time();

        printf("FPS: %f\r", fps);
        fflush(stdout);  //since the \n character is not used, stdout must be manually flushed
    }

    return 0;
}