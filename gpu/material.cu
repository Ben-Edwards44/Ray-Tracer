#include "utils.cu"
#include <vector>


__host__ __device__ class Texture {
    //ideally, I would use inheritance and polymorphism. But virtual functions are weird with CUDA, so I'll just use one big class
    public:
        static const int COLOUR = 0;
        static const int GRADIENT = 1;
        static const int CHECKERBOARD = 2;
        static const int IMAGE = 3;

        int type;

        __host__ __device__ Texture() {}

        __host__ Texture(int texture_type) {
            type = texture_type;
        }

        //initialisers for each texture type
        __host__ static Texture create_const_colour(Vec3 texture_colour) {
            Texture tex(COLOUR);
            tex.colour = texture_colour;

            return tex;
        }

        __host__ static Texture create_gradient() {
            return Texture(GRADIENT);
        }

        __host__ static Texture create_checkerboard(Vec3 light_colour, Vec3 dark_colour, int num_sq) {
            Texture tex(CHECKERBOARD);

            tex.light = light_colour;
            tex.dark = dark_colour;
            tex.num_squares = num_sq;

            return tex;
        }

        __host__ static Texture create_image(int width, int height, std::vector<Vec3> rgb_values) {
            Texture tex(IMAGE);

            tex.img_tex_width = width;
            tex.img_tex_height = height;

            tex.allocate_memory(rgb_values);

            return tex;
        }

        __device__ Vec3 get_texture_colour(Vec2 uv_coord) {
            float u = uv_coord.x;
            float v = uv_coord.y;

            switch (type) {
                case COLOUR:
                    return constant_colour();
                case GRADIENT:
                    return gradient(u, v);
                case CHECKERBOARD:
                    return checkerboard(u, v);
                case IMAGE:
                    return image(u, v);
                default:
                    return Vec3(0, 0, 0);
            }
        }

    private:
        //constant colour
        Vec3 colour;

        __device__ Vec3 constant_colour() {
            return colour;
        }

        //graident
        __device__ Vec3 gradient(float u, float v) {
            return Vec3(u, v, 0);
        }

        //checkerboard
        Vec3 light;
        Vec3 dark;

        int num_squares;

        __device__ Vec3 checkerboard(float u, float v) {
            int u_coord = u * num_squares;
            int v_coord = v * num_squares;

            if ((u_coord + v_coord) % 2 == 0) {
                return light;
            } else {
                return dark;
            }
        }

        //image
        int img_tex_width;
        int img_tex_height;

        Vec3 *img_rgb;

        __host__ void allocate_memory(std::vector<Vec3> rgb_values) {
            int size = sizeof(rgb_values[0]) * rgb_values.size();

            hipError_t error = hipMalloc((void **)&img_rgb, size);  //allocate the memory
            check_cuda_error(error);

            Vec3 *rgb_array = &rgb_values[0];  //get the pointer to the underlying array

            error = hipMemcpy(img_rgb, rgb_array, size, hipMemcpyHostToDevice);  //copy the data over
            check_cuda_error(error);
        }

        __device__ Vec3 image(float u, float v) {
            int u_coord = (img_tex_width - 1) * u;
            int v_coord = (img_tex_height - 1) * v;

            return img_rgb[v_coord * img_tex_width + u_coord];
        }
};


__host__ __device__ class Material {
    public:
        Texture texture;

        float smoothness;  //[0, 1]. 0 = perfect diffuse, 1 = perfect reflect
        float refractive_index;

        Vec3 emitted_light;

        bool need_uv;  //can optimise by not calculating uv coords if not needed
        bool is_glass = false;

        __host__ __device__ Material() {}

        __host__ Material(Texture mat_tex, float smoothness_val) {
            texture = mat_tex;
            smoothness = smoothness_val;

            emitted_light = Vec3(0, 0, 0);

            need_uv = texture.type != Texture::COLOUR;
        }

        __host__ Material(Texture mat_tex, float smoothness_val, float emit_strength, Vec3 emit_colour) {
            //constructor for emissive texture
            texture = mat_tex;
            smoothness = smoothness_val;
            emitted_light = emit_colour * emit_strength;

            need_uv = texture.type != Texture::COLOUR;
        }
};