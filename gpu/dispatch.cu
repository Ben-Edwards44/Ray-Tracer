#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <stdexcept>

#include "raytracer.cu"


void check_error(hipError_t error) {
    if (error != hipSuccess) {
        std::string err_msg = hipGetErrorString(error);
        throw std::runtime_error("Error from CUDA: " + err_msg);
    }
}


template<typename T>
class ReadWriteDeviceArray {
    public:
        T *array;

        ReadWriteDeviceArray(int array_len) {
            len = array_len;
            mem_size = len * sizeof(T);

            allocate_unified_mem();
        }

        void free_memory() {
            //the memory cannot be accessed by cpu or gpu after calling this
            hipFree(array);
        }

    private:
        int len;
        int mem_size;

        void allocate_unified_mem() {
            //allocate memory that can be accessed by both the gpu and cpu
            hipError_t error = hipMallocManaged(&array, mem_size);
            check_error(error);
        }
};


template <typename T>
class ReadOnlyDeviceArray {
    public:
        T *device_pointer;

        ReadOnlyDeviceArray(std::vector<T> values) {
            host_values = values;
            mem_size = sizeof(T) * values.size();

            allocate_mem();
        }

        void free_memory() {
            //should be called after we have finished with the data
            hipFree(device_pointer);
        }
    
    private:
        int mem_size;
        std::vector<T> host_values;

        void allocate_mem() {
            hipError_t error = hipMalloc((void **)&device_pointer, mem_size);  //allocate the memory
            check_error(error);
            
            T *host_array = &host_values[0];  //get the pointer to the underlying array

            error = hipMemcpy(device_pointer, host_array, mem_size, hipMemcpyHostToDevice);  //copy the value over
            check_error(error);
        }
};


template <typename T>
class ReadOnlyDeviceValue {
    public:
        T *host_value;
        T *device_pointer;

        ReadOnlyDeviceValue(T value) {
            host_value = &value;
            mem_size = sizeof(T);

            allocate_mem();
        }

        void free_memory() {
            //should be called after we have finished with the data
            hipFree(device_pointer);
        }
    
    private:
        int mem_size;

        void allocate_mem() {
            hipError_t error = hipMalloc((void **)&device_pointer, mem_size);  //allocate the memory
            check_error(error);
            
            error = hipMemcpy(device_pointer, host_value, mem_size, hipMemcpyHostToDevice);  //copy the value over
            check_error(error);
        }
};


class Scene {
    public:
        CamData cam_data;
        RenderData render_data;

        std::vector<Sphere> spheres;
        std::vector<Triangle> triangles;
        std::vector<Quad> quads;
        std::vector<OneWayQuad> one_way_quads;

        int len_pixel_array;

        std::vector<float> previous_render;

        AllMeshes all_mesh_struct;

        Scene(CamData cam, RenderData r_data, std::vector<Sphere> s, std::vector<Triangle> t, std::vector<Quad> q, std::vector<OneWayQuad> o_q, int len) {
            cam_data = cam;
            render_data = r_data;
            spheres = s;
            triangles = t;
            quads = q;
            one_way_quads = o_q;
            len_pixel_array = len;

            previous_render = std::vector<float>(len_pixel_array);

            all_mesh_struct = get_meshes();

            assign_constant_mem();
        }

    private:
        void assign_constant_mem() {
            //to be called before first scene (NOTE: no need to free constant memory)
            hipMemcpyToSymbol(HIP_SYMBOL(const_all_meshes), &all_mesh_struct, sizeof(all_mesh_struct));
            hipMemcpyToSymbol(HIP_SYMBOL(const_cam_data), &cam_data, sizeof(cam_data));
        }

        AllMeshes get_meshes() {
            //NOTE: I'm not sure I ever free the memory used here... (I'll just leave it for now)
            ReadOnlyDeviceArray<Sphere> d_spheres(spheres);
            ReadOnlyDeviceArray<Triangle> d_triangles(triangles);
            ReadOnlyDeviceArray<Quad> d_quads(quads);
            ReadOnlyDeviceArray<OneWayQuad> d_one_way_quads(one_way_quads);

            int num_spheres = spheres.size();
            int num_triangles = triangles.size();
            int num_quads = quads.size();
            int num_one_way_quads = one_way_quads.size();

            AllMeshes meshes{d_spheres.device_pointer, d_triangles.device_pointer, d_quads.device_pointer, d_one_way_quads.device_pointer, num_spheres, num_triangles, num_quads, num_one_way_quads};

            return meshes;
        }
};


dim3 get_block_size(int array_width, int array_height, dim3 thread_dim) {
    //we need to round up in cases where the array size is not divided exactly
    int blocks_x = array_width / thread_dim.x + 1;
    int blocks_y = array_height / thread_dim.y + 1;

    return dim3(blocks_x, blocks_y);
}


void run_ray_tracer(Scene *scene, int current_time_ms) {
    //run the raytacing script on the gpu and store the result in the data_obj previous_render
    //assign memory on the gpu 
    ReadOnlyDeviceValue<RenderData> r_data(scene->render_data);
    ReadOnlyDeviceValue<int> current_time(current_time_ms);

    ReadOnlyDeviceArray<float> prev_render(scene->previous_render);

    ReadWriteDeviceArray<float> image_pixels(scene->len_pixel_array);

    dim3 thread_dim(16, 16);  //max is 1024
    dim3 block_dim = get_block_size(scene->cam_data.image_width, scene->cam_data.image_height, thread_dim);

    get_pixel_colour<<<block_dim, thread_dim>>>(image_pixels.array, prev_render.device_pointer, r_data.device_pointer, current_time.device_pointer);  //launch kernel

    hipDeviceSynchronize();  //wait until gpu has finished

    //copy pixel data before freeing memory
    for (int i = 0; i < scene->len_pixel_array; i++) {
        scene->previous_render[i] = image_pixels.array[i];
    }

    //free memory
    r_data.free_memory();
    current_time.free_memory();
    prev_render.free_memory();
    image_pixels.free_memory();
}


void render(Scene *scene, int current_time_ms) {
    //run the ray tracer to render a scene and store the resulting pixel values in the previous_render in the scene object
    run_ray_tracer(scene, current_time_ms);  //result stored in the previous render
    scene->render_data.frame_num++;

    hipError_t error = hipPeekAtLastError();
    check_error(error);
}