#include "hip/hip_runtime.h"
#include "utils.cu"


const float INF = 100000;


__host__ __device__ struct CamData {
    //stored data needed by the device (calculated by the host)
    Vec3 pos;
    Vec3 tl_position;

    float focal_length;

    float delta_u;
    float delta_v;

    int image_width;
    int image_height;
};


__host__ __device__ struct RenderData {
    //data sent from host
    int rays_per_pixel;
    int reflection_limit;

    int frame_num;

    bool static_scene;

    Vec3 sky_colour;
};


__host__ __device__ struct Material {
    Vec3 colour;
    float emission_strength;
    Vec3 emission_colour;
    int mat_type;

    //optional
    float fuzz_level;
};


__device__ struct RayHitData {
    bool ray_hits;
    float ray_travelled_dist;
    Vec3 hit_point;
    Vec3 normal_vec;
};


__device__ class Ray {
    public:
        int pixel_x;
        int pixel_y;

        Vec3 origin;
        Vec3 direction;

        uint *rng_state;

        __device__ Ray(int p_x, int p_y, CamData *camera_data, uint *state) {
            pixel_x = p_x;
            pixel_y = p_y;

            rng_state = state;

            set_direction_origin(camera_data);
        }

        __device__ Vec3 get_pos(float dist) {
            return direction * dist + origin;
        }

        __device__ void reflect(RayHitData *hit_data, Material obj_material) {
            //reflect ray after hitting an object

            Vec3 new_direction;
            if (obj_material.mat_type == 0) {
                //diffuse_reflect(hit_data);
                new_direction = true_lambertian_reflect(hit_data);
            } else if (obj_material.mat_type == 1) {
                new_direction = perfect_reflect(hit_data);
            }
            else {
                new_direction = fuzzy_perfect_reflect(hit_data, obj_material);
            }

            direction = new_direction;
            origin = hit_data->hit_point;
        }
    
    private:
        __device__ Vec3 screen_to_world(int x, int y, CamData *camera_data) {
            //convert a point (x, y) on the viewport projection plane into a world space coordinate
            Vec3 local_pos;

            local_pos.x = x * camera_data->delta_u;
            local_pos.y = -y * camera_data->delta_v;
            local_pos.z = 0;

            return Vec3(camera_data->tl_position + local_pos);
        }

        __device__ void set_direction_origin(CamData *camera_data) {
            Vec3 view_pos = screen_to_world(pixel_x, pixel_y, camera_data);
            Vec3 o = camera_data->pos;
            Vec3 dir = view_pos - o;

            origin = o;
            direction = dir.normalised();
        }

        __device__ Vec3 diffuse_reflect(RayHitData *hit_data) {
            //diffuse reflect after hitting something (just choose a random direction)
            float dir_x = normally_dist_num(rng_state);
            float dir_y = normally_dist_num(rng_state);
            float dir_z = normally_dist_num(rng_state);

            Vec3 rand_vec(dir_x, dir_y, dir_z);

            if (rand_vec.dot(hit_data->normal_vec) < 0) {
                rand_vec = rand_vec * -1;  //invert since we want a vector that points outwards
            }

            return rand_vec.normalised();
        }

        __device__ Vec3 true_lambertian_reflect(RayHitData *hit_data) {
            //reflected vector proportional to cos of the angle
            Vec3 rand_offset_vec = diffuse_reflect(hit_data);
            Vec3 new_dir = hit_data->normal_vec + rand_offset_vec;

            return new_dir.normalised();
        }

        __device__ Vec3 perfect_reflect(RayHitData *hit_data) {
            //angle incidence = angle reflection: r=d−2(d⋅n)n (where d is incoming vector, n is normal and r in reflected)
            float dot = direction.dot(hit_data->normal_vec);
            Vec3 reflected_vec = direction - hit_data->normal_vec * 2 * dot;

            return reflected_vec.normalised();
        }

        __device__ Vec3 fuzzy_perfect_reflect(RayHitData *hit_data, Material obj_material) {
            //angle reflection = angle incidence + some noise
            Vec3 reflected_vec = perfect_reflect(hit_data);
            Vec3 rand_offset_vec = diffuse_reflect(hit_data) * obj_material.fuzz_level;

            Vec3 new_dir = reflected_vec + rand_offset_vec;

            return new_dir.normalised();
        }
};


struct Plane {
    //ax + by + cz + d = 0
    float a;
    float b;
    float c;
    float d;
};


__host__ __device__ class Sphere {
    public:
        Vec3 center;
        float radius;

        Material material;

        __host__ Sphere(Vec3 cent, float r, Material mat) {
            center = cent;
            radius = r;
            material = mat;
        };

        __device__ Sphere() {};

        __device__ RayHitData hit(Ray *ray) {
            //ray-sphere intersection results in quadratic equation t^2(d⋅d)−2td⋅(C−Q)+(C−Q)⋅(C−Q)−r^2=0
            //so we solve with quadratic formula
            Vec3 c_min_q = center - ray->origin;

            float a = ray->direction.dot(ray->direction);
            float b = ray->direction.dot(c_min_q) * (-2);
            float c = c_min_q.dot(c_min_q) - radius * radius;

            float discriminant = b * b - 4 * a * c;

            RayHitData hit_data;
            if (discriminant >= 0) {
                float ray_dist = (-b - sqrt(discriminant)) / (2 * a);  //negative solution to equation

                //only render spheres in front of camera
                if (ray_dist >= 0) {
                    Vec3 hit_point = ray->get_pos(ray_dist);

                    hit_data.ray_hits = true;
                    hit_data.ray_travelled_dist = ray_dist;
                    hit_data.hit_point = hit_point;
                    hit_data.normal_vec = (hit_point - center).normalised();  //vector pointing from center to point of intersection
                }
            } else {
                hit_data.ray_hits = false;
                hit_data.ray_travelled_dist = INF;
            }

            return hit_data;
        }
};


__host__ __device__ class Triangle {
    public:
        Material material;
        Vec3 points[3];

        __host__ __device__ Triangle(Vec3 point1, Vec3 point2, Vec3 point3, Material mat) {
            points[0] = point1;
            points[1] = point2;
            points[2] = point3;

            material = mat;

            precompute();
        }

        __device__ RayHitData hit(Ray *ray) {
            Vec3 vecs_to_corner[3];

            for (int i = 0; i < 3; i++) {
                vecs_to_corner[i] = points[i] - ray->origin;
            }

            int num_outside = 0;
            for (int i = 0; i < 3; i++) {
                Vec3 normal = vecs_to_corner[i].cross(vecs_to_corner[(i + 1) % 3]);
                float dot_prod = normal.dot(ray->direction);

                if (dot_prod < 0) {num_outside++;}  //ray points outside of the triangle
            }

            //depending on the order of the points, the normals to the sides always point in or alyawys point out. Therefore, a ray interects <=> it is always within the vecs or always outside the vecs
            bool ray_hits = num_outside == 0 || num_outside == 3;

            RayHitData hit_data;
            hit_data.ray_hits = ray_hits;

            if (ray_hits) {
                hit_data.ray_travelled_dist = get_ray_travelled_dist(ray);
                hit_data.hit_point = ray->get_pos(hit_data.ray_travelled_dist);

                Vec3 normal = normal_vec;
                if (normal.dot(ray->direction) < 0) {normal = normal * -1;}  //normal should point in same direction as the ray

                hit_data.normal_vec = normal;
            } else {
                hit_data.ray_travelled_dist = INF;
            }

            return hit_data;
        }

    private:
        Plane plane;
        Vec3 normal_vec;

        __host__ __device__ void precompute() {
            //precompute the plane the triangle lies on and (one of) its normal vectors. https://math.stackexchange.com/questions/2686606/equation-of-a-plane-passing-through-3-points
            Vec3 side1 = points[0] - points[1];
            Vec3 side2 = points[1] - points[2];

            normal_vec = side1.cross(side2).normalised();

            plane = {normal_vec.x, normal_vec.y, normal_vec.z};
            plane.d = -(plane.a * points[0].x + plane.b * points[0].y + plane.c * points[0].z);  //sub in a point to find constant
        }

        __device__ float get_ray_travelled_dist(Ray *ray) {
            //this algebra was worked out on paper (it is just the interesction between a line and plane really)
            float numerator = plane.d + plane.a * ray->origin.x + plane.b * ray->origin.y + plane.c * ray->origin.z;
            float denominator = plane.a * ray->direction.x + plane.b * ray->direction.y + plane.c * ray->direction.z;

            return -numerator / denominator;
        }
};


__device__ struct AllMeshes {
    Sphere *spheres;
    Triangle *triangles;

    int num_spheres;
    int num_triangles;
};


__device__ struct RayCollision {
    RayHitData hit_data;
    Material hit_mesh_material;
};


template <typename T>
__device__ RayCollision get_specific_mesh_collision(Ray *ray, T *meshes, int num_meshes) {
    //get the closest collision with a specific mesh (e.g. sphere, triangle). NOTE: error occurs if there are no meshes
    RayHitData hit_data;
    Material hit_mesh_material;

    //in the case where no collisions are found, the hit_data struct may have nonsense default values. So we set sensible ones here
    hit_data.ray_hits = false;
    hit_data.ray_travelled_dist = INF;

    for (int i = 0; i < num_meshes; i++) {
        RayHitData current_hit = meshes[i].hit(ray);

        if (!current_hit.ray_hits) {continue;}

        bool closest_to_cam = current_hit.ray_travelled_dist <= hit_data.ray_travelled_dist;  //is this the closest to the camera so far?
        bool precision_error = -0.001 < current_hit.ray_travelled_dist < 0.001;  //floating point errors can cause a reflected ray to intersect with the same object twice (its origin is put just inside the object)
        
        if (closest_to_cam && !precision_error)  {
            hit_data = current_hit;
            hit_mesh_material = meshes[i].material;
        }
    }

    return RayCollision{hit_data, hit_mesh_material};
}

__device__ RayCollision get_ray_collision(Ray *ray, AllMeshes *meshes) {
    RayCollision triangle_collision = get_specific_mesh_collision<Triangle>(ray, meshes->triangles, meshes->num_triangles);
    RayCollision sphere_collision = get_specific_mesh_collision<Sphere>(ray, meshes->spheres, meshes->num_spheres);

    if (triangle_collision.hit_data.ray_hits && triangle_collision.hit_data.ray_travelled_dist < sphere_collision.hit_data.ray_travelled_dist) {
        return triangle_collision;
    } else {
        return sphere_collision;
    }
}


__device__ Vec3 trace_ray(Ray *ray, AllMeshes *meshes, RenderData *render_data) {
    Vec3 final_colour(0, 0, 0);
    Vec3 current_ray_colour(1, 1, 1);

    for (int _ = 0; _ < render_data->reflection_limit; _++) {
        RayCollision collision = get_ray_collision(ray, meshes);

        if (!collision.hit_data.ray_hits) {
            //ray has not hit anything - it has hit sky
            final_colour = final_colour + render_data->sky_colour * current_ray_colour;
            break;
        }

        //final_colour = collision.hit_mesh_material.colour;
        //break;

        ray->reflect(&collision.hit_data, collision.hit_mesh_material);

        Material material = collision.hit_mesh_material;
        Vec3 mat_emitted_light = material.emission_colour * material.emission_strength;  //TODO: precalculate

        final_colour = final_colour + mat_emitted_light * current_ray_colour;
        current_ray_colour = current_ray_colour * material.colour;
    }

    return final_colour;
}


__device__ Vec3 get_ray_colour(Vec3 previous_colour, Ray ray, AllMeshes *meshes, RenderData *render_data) {
    Vec3 colour(0, 0, 0);

    for (int _ = 0; _ < render_data->rays_per_pixel; _++) {
        Ray ray_copy = ray;
        Vec3 ray_colour = trace_ray(&ray_copy, meshes, render_data);
        colour = colour + ray_colour;
    }

    colour = colour / render_data->rays_per_pixel;

    if (render_data->static_scene && render_data->frame_num > 0) {
        //use progressive rendering (take average of previous renders)
        Vec3 previous_sum = previous_colour * render_data->frame_num;
        return (colour + previous_sum) / (render_data->frame_num + 1);
    } else {
        return colour;
    }
}


__global__ void get_pixel_colour(float *pixel_array, float *previous_render, CamData *camera_data, AllMeshes *mesh_data, RenderData *render_data, int *current_time) {
    //TODO: the number of params in this function is simply obscene: use a struct to clean things up
    int pixel_coord_x = threadIdx.x + blockIdx.x * blockDim.x;
    int pixel_coord_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (pixel_coord_x >= camera_data->image_width || pixel_coord_y >= camera_data->image_height) {return;}  //account for grid size being too big
    
    int array_index = (pixel_coord_y * camera_data->image_width + pixel_coord_x) * 3;  //multiply by 3 to account for each pixel having r, b, g values

    Vec3 previous_colour(previous_render[array_index], previous_render[array_index + 1], previous_render[array_index + 2]);

    uint rng_state = array_index * 3145739 + *current_time * 6291469;

    Ray ray(pixel_coord_x, pixel_coord_y, camera_data, &rng_state);

    Vec3 colour = get_ray_colour(previous_colour, ray, mesh_data, render_data);

    pixel_array[array_index] = colour.x;
    pixel_array[array_index + 1] = colour.y;
    pixel_array[array_index + 2] = colour.z;
}