
#include <hip/hip_runtime.h>
#include <cmath>


__device__ float3 operator+(float3 &a, float3 &b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}


__device__ float3 operator-(float3 &a, float3 &b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}


__host__ __device__ struct CamData {
    //stored data needed by the device (calculated by the host)
    float3 pos;
    float3 tl_position;

    float focal_length;

    float delta_u;
    float delta_v;

    int image_width;
    int image_height;
};


__host__ __device__ class Vec3 {
    public:
        float x;
        float y;
        float z;

        __host__ __device__ Vec3(float val_x, float val_y, float val_z) {
            x = val_x;
            y = val_y;
            z = val_z;
        }

        __device__ Vec3(float3 vector) {
            x = vector.x;
            y = vector.y;
            z = vector.z;
        }

        __device__ Vec3() {}

        __device__ Vec3 operator+(Vec3 other_vec) {
            return Vec3(x + other_vec.x, y + other_vec.y, z + other_vec.z);
        }

        __device__ Vec3 operator-(Vec3 other_vec) {
            return Vec3(x - other_vec.x, y - other_vec.y, z - other_vec.z);
        }

        __device__ Vec3 operator*(float scalar) {
            return Vec3(x * scalar, y * scalar, z * scalar);
        }

        __device__ float magnitude() {
            float mag_sq = x * x + y * y + z * z;
            return sqrt(mag_sq);
        }

        __device__ Vec3 normalised() {
            float mag = magnitude();
            Vec3 unit_vec(x / mag, y / mag, z / mag);

            return unit_vec;
        }

        __device__ float dot(Vec3 other_vec) {
            float new_x = x * other_vec.x;
            float new_y = y * other_vec.y;
            float new_z = z * other_vec.z;

            return new_x + new_y + new_z;
        }
};


__device__ class Ray {
    public:
        int pixel_x;
        int pixel_y;

        Vec3 origin;
        Vec3 direction;

        __device__ Ray(int p_x, int p_y, CamData *camera_data) {
            pixel_x = p_x;
            pixel_y = p_y;

            set_direction_origin(camera_data);
        }

        __device__ Vec3 get_pos(float dist) {
            return origin + direction * dist;
        }

    private:
        __device__ Vec3 screen_to_world(int x, int y, CamData *camera_data) {
            //convert a point (x, y) on the viewport projection plane into a world space coordinate
            float3 local_pos;

            local_pos.x = x * camera_data->delta_u;
            local_pos.y = -y * camera_data->delta_v;
            local_pos.z = 0;

            return Vec3(camera_data->tl_position + local_pos);
        }

        __device__ void set_direction_origin(CamData *camera_data) {
            Vec3 view_pos = screen_to_world(pixel_x, pixel_y, camera_data);
            Vec3 o = camera_data->pos;
            Vec3 dir = view_pos - o;

            origin = o;
            direction = dir.normalised();
        }
};


__device__ struct RayHitData {
    bool ray_hits = false;
    float ray_travelled_dist = INFINITY;
    Vec3 hit_point;
    Vec3 normal_vec;
};


__host__ __device__ class Sphere {
    public:
        Vec3 center;
        float radius;

        Vec3 colour;

        __device__ RayHitData hit(Ray *ray) {
            //ray-sphere intersection results in quadratic equation t^2(d⋅d)−2td⋅(C−Q)+(C−Q)⋅(C−Q)−r^2=0
            //so we solve with quadratic formula
            Vec3 c_min_q = center - ray->origin;

            float a = ray->direction.dot(ray->direction);
            float b = ray->direction.dot(c_min_q) * (-2);
            float c = c_min_q.dot(c_min_q) - radius * radius;

            float discriminant = b * b - 4 * a * c;

            RayHitData hit_data;
            if (discriminant >= 0) {
                float ray_dist = (-b - sqrt(discriminant)) / (2 * a);  //negative solution to equation

                //only render spheres in front of camera
                if (ray_dist >= 0) {
                    Vec3 hit_point = ray->get_pos(ray_dist);

                    hit_data.ray_hits = true;
                    hit_data.ray_travelled_dist = ray_dist;
                    hit_data.hit_point = hit_point;
                    hit_data.normal_vec = (hit_point - center).normalised();  //vector pointing from center to point of intersection
                }
            }

            return hit_data;
        }
};


__device__ Vec3 get_ray_colour(Ray *ray, Sphere *mesh_data, int *num_spheres) {
    //check sphere intersection
    RayHitData hit_data;
    Sphere hit_sphere;
    for (int i = 0; i < *num_spheres; i++) {
        RayHitData current_hit = mesh_data[i].hit(ray);
        
        //check if this sphere is closest to camera
        if (current_hit.ray_travelled_dist <= hit_data.ray_travelled_dist) {
            hit_data = current_hit;
            hit_sphere = mesh_data[i];
        }
    }

    if (hit_data.ray_hits) {
        return hit_sphere.colour;
    } else {
        return Vec3(0, 0, 0);
    }
}


__global__ void get_pixel_colour(float *pixel_array, CamData *camera_data, Sphere *mesh_data, int *num_spheres) {
    int pixel_coord_x = threadIdx.x + blockIdx.x * blockDim.x;
    int pixel_coord_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (pixel_coord_x >= camera_data->image_width || pixel_coord_y >= camera_data->image_height) {return;}  //account for grid size being too big
    
    int array_index = (pixel_coord_y * camera_data->image_width + pixel_coord_x) * 3;  //multiply by 3 to account for each pixel having r, b, g values
    
    Ray ray(pixel_coord_x, pixel_coord_y, camera_data);

    Vec3 colour = get_ray_colour(&ray, mesh_data, num_spheres);

    pixel_array[array_index] = colour.x;
    pixel_array[array_index + 1] = colour.y;
    pixel_array[array_index + 2] = colour.z;
}