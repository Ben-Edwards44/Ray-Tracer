#include "hip/hip_runtime.h"
#include "utils.cu"


__host__ __device__ struct CamData {
    //stored data needed by the device (calculated by the host)
    Vec3 pos;
    Vec3 tl_position;

    float focal_length;

    float delta_u;
    float delta_v;

    int image_width;
    int image_height;
};


__host__ __device__ struct RenderData {
    //data sent from host
    int rays_per_pixel;
    int reflection_limit;

    int num_spheres;

    int frame_num;

    bool static_scene;
};


__device__ struct RayHitData {
    bool ray_hits = false;
    float ray_travelled_dist = INFINITY;
    Vec3 hit_point;
    Vec3 normal_vec;
};


__device__ class Ray {
    public:
        int pixel_x;
        int pixel_y;

        Vec3 origin;
        Vec3 direction;

        uint *rng_state;

        __device__ Ray(int p_x, int p_y, CamData *camera_data, uint *state) {
            pixel_x = p_x;
            pixel_y = p_y;

            rng_state = state;

            set_direction_origin(camera_data);
        }

        __device__ Vec3 get_pos(float dist) {
            return direction * dist + origin;
        }

        __device__ void reflect(RayHitData *hit_data) {
            //reflect ray after hitting an object

            //TODO: decide whether to use diffuse or perfect
            //diffuse_reflect(hit_data);
            //perfect_reflect(hit_data);
            true_lambertian_reflect(hit_data);

            origin = hit_data->hit_point;
        }
    private:
        __device__ Vec3 screen_to_world(int x, int y, CamData *camera_data) {
            //convert a point (x, y) on the viewport projection plane into a world space coordinate
            Vec3 local_pos;

            local_pos.x = x * camera_data->delta_u;
            local_pos.y = -y * camera_data->delta_v;
            local_pos.z = 0;

            return Vec3(camera_data->tl_position + local_pos);
        }

        __device__ void set_direction_origin(CamData *camera_data) {
            Vec3 view_pos = screen_to_world(pixel_x, pixel_y, camera_data);
            Vec3 o = camera_data->pos;
            Vec3 dir = view_pos - o;

            origin = o;
            direction = dir.normalised();
        }

        __device__ void diffuse_reflect(RayHitData *hit_data) {
            //diffuse reflect after hitting something
            float dir_x = normally_dist_num(rng_state);
            float dir_y = normally_dist_num(rng_state);
            float dir_z = normally_dist_num(rng_state);

            Vec3 new_dir(dir_x, dir_y, dir_z);

            if (new_dir.dot(hit_data->normal_vec) < 0) {
                new_dir = new_dir * -1;  //invert since we are reflecting inside the sphere
            }

            direction = new_dir.normalised();
        }

        __device__ void perfect_reflect(RayHitData *hit_data) {
            //angle incidence = angle reflection: r=d−2(d⋅n)n (where d is incoming vector, n is normal and r in reflected)
            float dot = direction.dot(hit_data->normal_vec);
            Vec3 reflected_vec = direction - hit_data->normal_vec * 2 * dot;

            direction = reflected_vec;
        }

        __device__ void true_lambertian_reflect(RayHitData *hit_data) {
            //reflected vector proportional to cos of the angle
            float dir_x = normally_dist_num(rng_state);
            float dir_y = normally_dist_num(rng_state);
            float dir_z = normally_dist_num(rng_state);

            Vec3 rand_dir(dir_x, dir_y, dir_z);

            if (rand_dir.dot(hit_data->normal_vec) < 0) {
                rand_dir = rand_dir * -1;  //invert since we are reflecting inside the sphere
            }

            Vec3 new_dir = hit_data->normal_vec + rand_dir;

            direction = new_dir.normalised();
        }
};


__host__ __device__ struct Material {
    Vec3 colour;
    float emission_strength;
    Vec3 emission_colour;
};


__host__ __device__ class Sphere {
    public:
        Vec3 center;
        float radius;

        Material material;

        __host__ Sphere(Vec3 cent, float r, Material mat) {
            center = cent;
            radius = r;
            material = mat;
        };

        __device__ Sphere() {};

        __device__ RayHitData hit(Ray *ray) {
            //ray-sphere intersection results in quadratic equation t^2(d⋅d)−2td⋅(C−Q)+(C−Q)⋅(C−Q)−r^2=0
            //so we solve with quadratic formula
            Vec3 c_min_q = center - ray->origin;

            float a = ray->direction.dot(ray->direction);
            float b = ray->direction.dot(c_min_q) * (-2);
            float c = c_min_q.dot(c_min_q) - radius * radius;

            float discriminant = b * b - 4 * a * c;

            RayHitData hit_data;
            if (discriminant >= 0) {
                float ray_dist = (-b - sqrt(discriminant)) / (2 * a);  //negative solution to equation

                //only render spheres in front of camera
                if (ray_dist >= 0) {
                    Vec3 hit_point = ray->get_pos(ray_dist);

                    hit_data.ray_hits = true;
                    hit_data.ray_travelled_dist = ray_dist;
                    hit_data.hit_point = hit_point;
                    hit_data.normal_vec = (hit_point - center).normalised();  //vector pointing from center to point of intersection
                }
            }

            return hit_data;
        }
};


__device__ struct RayCollision {
    RayHitData *hit_data;
    Sphere *hit_sphere;
};


__device__ RayCollision get_ray_collision(Ray *ray, Sphere *mesh_data, int num_spheres) {
    RayHitData hit_data;
    Sphere hit_sphere;

    for (int i = 0; i < num_spheres; i++) {
        RayHitData current_hit = mesh_data[i].hit(ray);

        if (!current_hit.ray_hits) {continue;}

        bool closest_to_cam = current_hit.ray_travelled_dist <= hit_data.ray_travelled_dist;  //is this the closest to the camera so far?
        bool precision_error = -0.001 < current_hit.ray_travelled_dist < 0.001;  //floating point errors can cause a reflected ray to intersect with the same object twice (its origin is put just inside the object)
        
        if (closest_to_cam && !precision_error)  {
            hit_data = current_hit;
            hit_sphere = mesh_data[i];
        }
    }

    return RayCollision{&hit_data, &hit_sphere};
}


__device__ Vec3 trace_ray(Ray *ray, Sphere *mesh_data, RenderData *render_data) {
    Vec3 final_colour(0, 0, 0);
    Vec3 current_ray_colour(1, 1, 1);

    for (int _ = 0; _ < render_data->reflection_limit; _++) {
        RayCollision collision = get_ray_collision(ray, mesh_data, render_data->num_spheres);

        if (!collision.hit_data->ray_hits) {
            //ray has not hit anything - sky
            //Vec3 sky_light(0.6, 0.6, 0.8);
            //final_colour = final_colour + sky_light * current_ray_colour;

            break;
        }

        ray->reflect(collision.hit_data);

        Material material = collision.hit_sphere->material;
        Vec3 mat_emitted_light = material.emission_colour * material.emission_strength;  //TODO: precalculate

        final_colour = final_colour + mat_emitted_light * current_ray_colour;
        current_ray_colour = current_ray_colour * material.colour;
    }

    return final_colour;
}


__device__ Vec3 get_ray_colour(Vec3 previous_colour, Ray ray, Sphere *mesh_data, RenderData *render_data) {
    //check sphere intersection
    Vec3 colour(0, 0, 0);

    for (int _ = 0; _ < render_data->rays_per_pixel; _++) {
        Ray ray_copy = ray;
        Vec3 ray_colour = trace_ray(&ray_copy, mesh_data, render_data);
        colour = colour + ray_colour;
    }

    colour = colour / render_data->rays_per_pixel;

    if (render_data->static_scene && render_data->frame_num > 0) {
        //use progressive rendering (take average of previous renders)
        Vec3 previous_sum = previous_colour * render_data->frame_num;
        return (colour + previous_sum) / (render_data->frame_num + 1);
    } else {
        return colour;
    }
}


__global__ void get_pixel_colour(float *pixel_array, float *previous_render, CamData *camera_data, Sphere *mesh_data, RenderData *render_data, int *current_time) {
    //TODO: the number of params in this function is simply obscene: use a struct to clean things up
    
    int pixel_coord_x = threadIdx.x + blockIdx.x * blockDim.x;
    int pixel_coord_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (pixel_coord_x >= camera_data->image_width || pixel_coord_y >= camera_data->image_height) {return;}  //account for grid size being too big
    
    int array_index = (pixel_coord_y * camera_data->image_width + pixel_coord_x) * 3;  //multiply by 3 to account for each pixel having r, b, g values

    Vec3 previous_colour(previous_render[array_index], previous_render[array_index + 1], previous_render[array_index + 2]);

    uint rng_state = array_index * 3145739 + *current_time * 6291469;

    Ray ray(pixel_coord_x, pixel_coord_y, camera_data, &rng_state);

    Vec3 colour = get_ray_colour(previous_colour, ray, mesh_data, render_data);

    pixel_array[array_index] = colour.x;
    pixel_array[array_index + 1] = colour.y;
    pixel_array[array_index + 2] = colour.z;
}